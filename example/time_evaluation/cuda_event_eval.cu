#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <time.h>

int main() {
    hipEvent_t start, stop;
    
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipStream_t stream;
    
    hipStreamCreate(&stream);
    
    hipEventRecord(start, stream);
    
    sleep(2);
    hipEventRecord(stop, stream);
    
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time elapsed: %f ms\n", milliseconds);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipStreamDestroy(stream);
    
    return 0;
}
