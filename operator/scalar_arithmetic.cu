// scalar_add.cu

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void ScalarAdd(float* a, float* b, float* c)
{
    *c = *a + *b;
}
__global__ void ScalarMultiply(float* a, float* b, float* c)
{
    *c = *a * *b;
}

int main()
{
    // Host variables
    float h_a = 1.0f;
    float h_b = 2.7f;
    float h_c = 0.0f;

    // Device variables
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, sizeof(float));
    hipMalloc(&d_b, sizeof(float));
    hipMalloc(&d_c, sizeof(float));

    hipMemcpy(d_a, &h_a, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &h_b, sizeof(float), hipMemcpyHostToDevice);

    ScalarAdd<<<1, 1>>>(d_a, d_b, d_c);
    hipMemcpy(&h_c, d_c, sizeof(float), hipMemcpyDeviceToHost);
    printf("%.1f + %.1f = %.1f\n", h_a, h_b, h_c);

    ScalarMultiply<<<1, 1>>>(d_a, d_b, d_c);
    hipMemcpy(&h_c, d_c, sizeof(float), hipMemcpyDeviceToHost);
    printf("%.1f * %.1f = %.1f\n", h_a, h_b, h_c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
