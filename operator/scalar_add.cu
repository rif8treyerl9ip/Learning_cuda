// scalar_add.cu

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void ScalarAdd(float* a, float* b, float* c)
{
    *c = *a + *b;
}

int main()
{
    // Host variables
    float h_a = 1.5f;
    float h_b = 2.7f;
    float h_c = 0.0f;

    // Device variables
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, sizeof(float));
    hipMalloc(&d_b, sizeof(float));
    hipMalloc(&d_c, sizeof(float));

    // Copy input data to device
    hipMemcpy(d_a, &h_a, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &h_b, sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    ScalarAdd<<<1, 1>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(&h_c, d_c, sizeof(float), hipMemcpyDeviceToHost);

    // Print result
    printf("%.1f + %.1f = %.1f\n", h_a, h_b, h_c);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
